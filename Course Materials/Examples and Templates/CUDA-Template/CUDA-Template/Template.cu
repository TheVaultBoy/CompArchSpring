#include "hip/hip_runtime.h"
#include <stdio.h>
#include <conio.h>
#include <math.h>
#include <cutil.h>
#include <hip/hip_runtime_api.h>
#include <algorithm>
using namespace std;
#pragma comment(lib, "cutil32D.lib")

int rint(int l, int h)
{
	int r = h-l+1, res;
	res = rand()%r+l;
	return res;
}

int* rintarr(int size, int l, int h)
{
	int *res = new int[size];//, t;
	for (int i =0; i <= size - 1; i++)
	{	
		//t = rint(0, size);
		res[i] = size-i;//rint(l+t*h, h+t*h);
	}
	return res;
}

void iaprint(int *arr, int size)
{
	for (int i =0; i <= size - 1; i++)
		printf("%i ", arr[i]);
	printf("\n");
}

//������� ���������� �� �����������
hipDeviceProp_t* ShowDeviceInfo()
{
	int deviceCount;
	hipDeviceProp_t deviceProp;
	
	//�������� � ������� ���������� ���������
	hipGetDeviceCount(&deviceCount);
	printf("Device count: %d\n\n", deviceCount);

	for (int i = 0; i <= deviceCount - 1; i++)
	{
		//�������� ���������� � ������� ����������
		hipGetDeviceProperties(&deviceProp, i);
		
		//������� ��������� �� ����������
		printf("Device name: %s\n", deviceProp.name);
		printf("Total global memory: %d\n", deviceProp.totalGlobalMem);
		printf("Shared memory per block: %d\n", deviceProp.sharedMemPerBlock);
		printf("Registers per block: %d\n", deviceProp.regsPerBlock);
		printf("Warp size: %d\n", deviceProp.warpSize);
		printf("Memory pitch: %d\n", deviceProp.memPitch);
		printf("Max threads per block: %d\n", deviceProp.maxThreadsPerBlock);
		printf("Max threads dimensions: x = %d, y = %d, z = %d\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
		printf("Max grid size: x = %d, y = %d, z = %d\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]); 
		printf("Clock rate: %d\n", deviceProp.clockRate);
		printf("Total constant memory: %d\n", deviceProp.totalConstMem);
		printf("Compute capability: %d.%d\n", deviceProp.major, deviceProp.minor);
		printf("Texture alignment: %d\n", deviceProp.textureAlignment);
		printf("Device overlap: %d\n", deviceProp.deviceOverlap);
		printf("Multiprocessor count: %d\n", deviceProp.multiProcessorCount);
		printf("Kernel execution timeout enabled: %s\n", deviceProp.kernelExecTimeoutEnabled ? "true" : "false");
	}
	
	return &deviceProp;
}

//������������� ������: global - GPU �� CPU; device - GPU �� GPU; host - CPU �� CPU
//����������� ���� ����
__global__ void vadd(float *v1, float *v2, float *v3)
{
	int cur = threadIdx.x; //��������� ������� ������� ����
	v3[cur] = v1[cur] + v2[cur];
}

__global__ void sort(int *arr, int size)
{
	int i = threadIdx.x;
	int k = 0, b = arr[i];
		for (int j = 0; j <= size - 1; j++)
			if (arr[j] < b) k++;
	__syncthreads();
	arr[k] = b;
}

int main()
{
	/*--------------------vadd---------------------*/
	unsigned int t1;
	cutCreateTimer(&t1);
	cutResetTimer(t1);
	hipSetDevice(0);
	
	hipDeviceProp_t *deviceProp = ShowDeviceInfo();
	
	int size = deviceProp->maxThreadsPerBlock;
	float *v1 = new float[size];
	float *v2 = new float[size];
	float *v3 = new float[size];
	for (int i = 0; i <= size - 1; i++)
	{
		v1[i] = (float)i/10 + (float)i;
		v2[i] = (float)i/5 + (float)2*i;
	}
	
	float *cv1, *cv2, *cv3;
	hipMalloc((void**)&cv1, size*sizeof(float));
	hipMalloc((void**)&cv2, size*sizeof(float));
	hipMalloc((void**)&cv3, size*sizeof(float));
	hipMemcpy(cv1, v1, size*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cv2, v2, size*sizeof(float), hipMemcpyHostToDevice);
	
	dim3 gridSize = dim3(1, 1, 1); //���������� ������ ���� ����
	dim3 blockSize = dim3(size, 1, 1); //���������� size �����
	//������� ��������� ��������
	hipEvent_t event;
	hipEventCreate(&event);
	hipEventRecord(event, 0); //0 - ����� ��������� ������
	cutStartTimer(t1);
	vadd<<<gridSize, blockSize>>>(cv1, cv2, cv3); //���������� ������� (����������)
	hipEventSynchronize(event); //�������� ��������� ������ ���� ����� GPU
	cutStopTimer(t1);
	
	hipMemcpy(v3, cv3, size*sizeof(float), hipMemcpyDeviceToHost);
	/*for (int i = 0; i <= size - 1; i++)
	{
		printf("%.2f + %.2f = %.2f\n", v1[i], v2[i], v3[i]);
	}*/
	printf("\nGPU vadd: %f ms\n\n", cutGetTimerValue(t1));
	
	hipEventDestroy(event);
	hipFree(cv1); hipFree(cv2); hipFree(cv3);
	delete[] v1; delete[] v2; delete[] v3;
	getch();
	/*---------------------------------------------*/
	
	/*--------------------sort---------------------*/
	srand(0);
	int s = 100;
	int *arr = rintarr(s, 0, 5);
	iaprint(arr, s);
	printf("\n");
	
	int *carr;
	hipMalloc((void**)&carr, s*sizeof(int));
	hipEvent_t e;
	hipEventCreate(&e);
	hipEventRecord(e, 0);
	cutStartTimer(t1);
	hipMemcpy(carr, arr, s*sizeof(int), hipMemcpyHostToDevice);
	sort<<<1, s>>>(carr, s);
	hipEventSynchronize(e);
	cutStopTimer(t1);
	
	hipEventDestroy(e);
	int *brr = new int[s];
	hipMemcpy(brr, carr, s*sizeof(int), hipMemcpyDeviceToHost);
	iaprint(brr, s);
	printf("GPU vadd: %f ms\n\n", cutGetTimerValue(t1));
	
	cutStartTimer(t1);
	sort(arr, arr+s);
	iaprint(arr, s);
	cutStopTimer(t1);
	printf("CPU vadd: %f ms\n\n", cutGetTimerValue(t1));
	
	hipFree(carr);
	delete[] arr;
	delete[] brr;
	getch();
	/*---------------------------------------------*/
	return 0;
}