#include "hip/hip_runtime.h"
#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "Header.h"

#define TX 27 // number of threads per block along x-axis
#define TY 27 // number of threads per block along y-axis

__global__ void greyscaleKernel(pixel* img, unsigned W, unsigned H, unsigned dim)
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;

  unsigned int r_avg = 0;
  unsigned int g_avg = 0;
  unsigned int b_avg = 0;
  int temp = (dim - 1) / 2;
  if ((((y - temp)*W + x - temp) < W*H) && (((y + temp)*W + x + temp) < W*H))
  {
    for (int i = temp; i > -(temp+1); i--)
    {
      for (int j = -temp; j < (temp+1); j++)
      {
        r_avg += img[(y + i)*W + x + j].r;
        g_avg += img[(y + i)*W + x + j].g;
        b_avg += img[(y + i)*W + x + j].b;
      }
    }
  }
  img[y*W + x].r = r_avg/(dim*dim);
  img[y*W + x].g = g_avg/(dim*dim);
  img[y*W + x].b = b_avg/ (dim*dim);
}


// Helper function for using CUDA to add vectors in parallel.
void cudaGrayscale(pixel* h_img, unsigned W, unsigned H, unsigned dim)
{
  pixel* d_img;
  size_t bytes = W * H * sizeof(pixel);
  hipMalloc(&d_img, bytes);
  hipMemcpy(d_img, h_img, bytes, hipMemcpyHostToDevice);

  dim3 blockSize(TX, TY);
  int bx = (W + 26) / blockSize.x;
  int by = (H + 26) / blockSize.y;
  dim3 gridSize = dim3(bx, by);

  // Launch a kernel on the GPU with one thread for each element.
  greyscaleKernel<<<gridSize, blockSize>>>(d_img, W, H ,dim);

  hipMemcpy(h_img, d_img, bytes, hipMemcpyDeviceToHost);
  hipFree(d_img);
}
